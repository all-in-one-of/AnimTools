#include "hip/hip_runtime.h"
// GPU anim test: evaluate RDFT-encoded motion data using CUDA
// Author: Sergey Chaban <sergey.chaban@gmail.com>

#include <hip/hip_runtime.h>
#include <math_functions.h>
#include <hip/hip_math_constants.h>

#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <omp.h>

#define WIN32_LEAN_AND_MEAN 1
#define NOMINMAX
#define _WIN32_WINNT 0x0601
#include <Windows.h>

int64_t timestamp() {
	LARGE_INTEGER ctr;
	::QueryPerformanceCounter(&ctr);
	return ctr.QuadPart;
}

//---------------------------------------------------------------
enum E_MOT_TRK { POS, ROT, SCL };
enum E_MOT_RORD { XYZ, XZY, YXZ, YZX, ZXY, ZYX };
enum E_MOT_XORD { SRT, STR, RST, RTS, TSR, TRS };

typedef char MOT_STRING[0x40];

union MOT_VEC {
	struct { float x, y, z; };
	float v[3];
};

union MOT_QUAT {
	struct { float x, y, z, w; };
	float v[4];
};

struct MOT_TRACK {
	MOT_VEC vmin;
	MOT_VEC vmax;
	uint8_t srcMask;
	uint8_t dataMask;
	uint8_t reserved[6];
};

struct MOT_NODE {
	MOT_STRING name;
	uint32_t offs[3];
	uint8_t xord;
	uint8_t rord;
	uint8_t reserved[2];
	MOT_TRACK trk[3];
};

struct MOT_CLIP {
	char sig[4];
	float fps;
	uint32_t nfrm;
	uint32_t nnod;
	MOT_STRING name;
	MOT_NODE nodes[1];
};

MOT_QUAT motQuatExp(MOT_VEC v) {
	MOT_QUAT q = {};
	float ha = sqrtf(v.x*v.x + v.y*v.y + v.z*v.z);
	float s = fabsf(ha) < 1.0e-4f ? 1.0f : sinf(ha) / ha;
	q.x = v.x * s;
	q.y = v.y * s;
	q.z = v.z * s;
	q.w = cosf(ha);
	s = 1.0f / sqrtf(q.x*q.x + q.y*q.y + q.z*q.z + q.w*q.w);
	q.x *= s;
	q.y *= s;
	q.z *= s;
	q.w *= s;
	return q;
}

bool motCkNodeIdx(const MOT_CLIP* pClip, int nodeIdx) { return pClip && ((uint32_t)nodeIdx < pClip->nnod); }

bool motCkFrameNo(const MOT_CLIP* pClip, int fno) { return pClip && ((uint32_t)fno < pClip->nfrm); }

float* motGetTrackData(const MOT_CLIP* pClip, int nodeIdx, E_MOT_TRK trk) {
	float* p = nullptr;
	if (pClip && motCkNodeIdx(pClip, nodeIdx)) {
		int itrk = (int)trk;
		if (itrk < 3) {
			uint32_t offs = pClip->nodes[nodeIdx].offs[itrk];
			if (offs) {
				char* pTop = (char*)pClip;
				p = (float*)&pTop[offs];
			}
		}
	}
	return p;
}

void motGetChanData(const MOT_CLIP* pClip, int nodeIdx, E_MOT_TRK trk, int chIdx, float** ppData, int* pStride) {
	int stride = 0;
	float* p = nullptr;
	if ((uint32_t)chIdx < 3) {
		float* pTrk = motGetTrackData(pClip, nodeIdx, trk);
		if (pTrk) {
			int dataMask = pClip->nodes[nodeIdx].trk[(int)trk].dataMask;
			for (int i = 0; i < 3; ++i) {
				if (dataMask & (1 << i)) {
					p = pTrk + stride;
					++stride;
				}
			}
		}
	}
	if (ppData) {
		*ppData = p;
	}
	if (pStride) {
		*pStride = stride;
	}
}

MOT_VEC motGetVec(const MOT_CLIP* pClip, int nodeIdx, int fno, E_MOT_TRK trk) {
	MOT_VEC v = {};
	if (pClip && motCkNodeIdx(pClip, nodeIdx) && motCkFrameNo(pClip, fno)) {
		float* p = motGetTrackData(pClip, nodeIdx, trk);
		if (p) {
			int itrk = (int)trk;
			if (itrk < 3) {
				float defVal = trk == SCL ? 1.0f : 0.0f;
				int dataMask = pClip->nodes[nodeIdx].trk[itrk].dataMask;
				int srcMask = pClip->nodes[nodeIdx].trk[itrk].srcMask;
				int vsize = 0;
				for (int i = 0; i < 3; ++i) {
					if (dataMask & (1 << i)) ++vsize;
				}
				p += fno * vsize;
				for (int i = 0; i < 3; ++i) {
					if (dataMask & (1 << i)) {
						v.v[i] = *p++;
					} else if (srcMask & (1 << i)) {
						v.v[i] = pClip->nodes[nodeIdx].trk[itrk].vmin.v[i];
					} else {
						v.v[i] = defVal;
					}
				}
			}
		}
	}
	return v;
}

MOT_QUAT motGetQuat(const MOT_CLIP* pClip, int nodeIdx, int fno) {
	return motQuatExp(motGetVec(pClip, nodeIdx, fno, ROT));
}

MOT_CLIP* motClipLoad(const char* pPath) {
	MOT_CLIP* pClip = nullptr;
	FILE* f = fopen(pPath, "rb");
	if (f) {
		long len = 0;
		if (0 == fseek(f, 0, SEEK_END)) {
			len = ftell(f);
		}
		fseek(f, 0, SEEK_SET);
		if (len) {
			pClip = (MOT_CLIP*)malloc(len);
			if (pClip) {
				fread(pClip, len, 1, f);
			}
		}
		fclose(f);
	}
	return pClip;
}

void motClipUnload(MOT_CLIP* pClip) {
	if (pClip) { free(pClip); }
}
//---------------------------------------------------------------

void RDFT_fwd(float* pDst, const float* pSrc, int nsrc, int stride) {
	int n = (nsrc & 1) ? nsrc + 1 : nsrc;
	int hn = n / 2;
	float* pRe = pDst;
	float* pIm = pDst + hn;
	float nrm = 1.0f / (float)hn;
	float s = atanf(1.0f) * 8.0f / (float)n;
	for (int i = 0; i < n; ++i) {
		pDst[i] = 0.0f;
	}
	for (int i = 0; i < hn; ++i) {
		for (int j = 0; j < n; ++j) {
			int idx = (j % nsrc) * stride;
			float val = pSrc[idx];
			float t = s * (float)i * (float)j;
			pRe[i] += val * cosf(t);
			pIm[i] -= val * sinf(t);
		}
	}
	for (int i = 0; i < hn; ++i) {
		pRe[i] *= nrm;
	}
	pRe[0] /= 2;
	pRe[hn - 1] /= 2;
	for (int i = 0; i < hn; ++i) {
		pIm[i] *= -nrm;
	}
}

void RDFT_inv(float* pDst, const float* pSrc, int ndst) {
	int n = (ndst & 1) ? ndst + 1 : ndst;
	int hn = n / 2;
	const float* pRe = pSrc;
	const float* pIm = pSrc + hn;
	float s = atanf(1.0f) * 8.0f / (float)n;
	for (int i = 0; i < ndst; ++i) {
		pDst[i] = 0.0f;
	}
	for (int i = 0; i < ndst; ++i) {
		float t = s * (float)i;
		for (int j = 0; j < hn; ++j) {
			float re = pRe[j];
			float im = pIm[j];
			float r = t * (float)j;
			pDst[i] += re*cosf(r) + im*sinf(r);
		}
	}
}

class cMotion {
public:
	struct ROT_CHANNEL {
		float* pData;
		int stride;
		int nodeId;
		int chId;
		float* pCoefs;
		int cut;
	};

protected:
	MOT_CLIP* mpClip;
	int mPosVecsNum;
	MOT_VEC* mpPosVecs;
	int mRotChansNum;
	ROT_CHANNEL* mpRotChans;
	int mCoefsNum;
	float mParamFactor;
	int mCut;
	float* mpEvalCoefsCPU;
	float* mpEvalCoefsDev;
	float* mpEvalResCPU;
	float* mpEvalResDev;
	hipEvent_t mEvt;

	void eval_pos_vecs(float frame);

public:
	cMotion()
	:
	mpClip(nullptr),
	mPosVecsNum(0), mpPosVecs(nullptr),
	mRotChansNum(0), mpRotChans(nullptr),
	mCoefsNum(0), mParamFactor(0.0f), mCut(0),
	mpEvalCoefsCPU(nullptr), mpEvalCoefsDev(nullptr),
	mpEvalResCPU(nullptr), mpEvalResDev(nullptr)
	{}

	~cMotion() {
		unload();
	}

	void load(const char* pPath);
	void unload();

	float frame_to_param(float frame) {
		if (!mpClip) return 0.0f;
		float n = (float)mpClip->nfrm;
		float f = ::fmodf(::fabsf(frame), n);
		return f * mParamFactor;
	}

	int get_nfrm() const { return mpClip ? mpClip->nfrm : 0; }
	int get_nrot() const { return mRotChansNum; }
	int get_npos() const { return mPosVecsNum; }
	int get_ncut() const { return mCut; }
	float* get_res_ptr() { return mpEvalResCPU; }

	void clear_res() {
		if (mpEvalResCPU) {
			::memset(mpEvalResCPU, 0, mRotChansNum * sizeof(float));
		}
	}

	void eval_cpu(float frame);
	void eval_dev(float frame);
};

void cMotion::load(const char* pPath) {
	mpClip = motClipLoad(pPath);
	if (!mpClip) return;
	int numNodes = mpClip->nnod;
	mPosVecsNum = 0;
	for (int i = 0; i < numNodes; ++i) {
		int srcMask = mpClip->nodes[i].trk[POS].dataMask;
		int dataMask = mpClip->nodes[i].trk[POS].dataMask;
		if (srcMask || dataMask) {
			++mPosVecsNum;
		}
	}
	if (mPosVecsNum > 0) {
		mpPosVecs = (MOT_VEC*)::malloc(mPosVecsNum * sizeof(MOT_VEC));
	}
	mRotChansNum = 0;
	for (int i = 0; i < numNodes; ++i) {
		int dataMask = mpClip->nodes[i].trk[ROT].dataMask;
		for (int j = 0; j < 3; ++j) {
			if (dataMask & (1 << j)) {
				++mRotChansNum;
			}
		}
	}
	if (mRotChansNum > 0) {
		mpRotChans = (ROT_CHANNEL*)::malloc(mRotChansNum * sizeof(ROT_CHANNEL));
	}
	if (!mpRotChans) return;
	int chIdx = 0;
	for (int i = 0; i < numNodes; ++i) {
		int dataMask = mpClip->nodes[i].trk[ROT].dataMask;
		for (int j = 0; j < 3; ++j) {
			if (dataMask & (1 << j)) {
				ROT_CHANNEL* pCh = &mpRotChans[chIdx++];
				pCh->nodeId = i;
				pCh->chId = j;
				motGetChanData(mpClip, i, ROT, j, &pCh->pData, &pCh->stride);
			}
		}
	}

	int nfrm = mpClip->nfrm;
	float* pTmp = 0 ? (float*)::malloc(nfrm * sizeof(float)) : nullptr;
	int ncoef = nfrm;
	if (ncoef & 1) ++ncoef;
	mCoefsNum = ncoef;
	mParamFactor = ::atanf(1.0f) * 8.0f / (float)ncoef;
	for (int i = 0; i < mRotChansNum; ++i) {
		ROT_CHANNEL* pCh = &mpRotChans[i];
		pCh->pCoefs = (float*)::malloc(ncoef * sizeof(float));
		if (pCh->pCoefs) {
			RDFT_fwd(pCh->pCoefs, pCh->pData, nfrm, pCh->stride);
			if (pTmp) {
				RDFT_inv(pTmp, pCh->pCoefs, nfrm);
				::printf("-- [%d] %s:%c\n", i, mpClip->nodes[pCh->nodeId].name, "xyz"[pCh->chId]);
				for (int k = 0; k < nfrm; ++k) {
					float ref = pCh->pData[k*pCh->stride];
					float val = pTmp[k];
					::printf("[%d]: %.4f - %.4f = %f\n", k, ref, val, ref - val);
				}
			}
		}
	}
	if (pTmp) {
		::free(pTmp);
		pTmp = nullptr;
	}
	int minCut = nfrm + 1;
	int maxCut = 0;
	for (int i = 0; i < mRotChansNum; ++i) {
		ROT_CHANNEL* pCh = &mpRotChans[i];
		pCh->cut = (nfrm & (~1)) / 2;
		float* pRe = pCh->pCoefs;
		if (pRe) {
			const float qs = 0.0005f;
			for (int j = 0; j < ncoef / 2; ++j) {
				float x = pRe[j];
				float qx = ::floorf(::fabsf(x) / qs) * qs * (x < 0.0f ? -1.0f : 1.0f);
				if (qx == 0) {
					pCh->cut = j;
					break;
				}
			}
		}
		if (pCh->cut < minCut) {
			minCut = pCh->cut;
		}
		if (pCh->cut > maxCut) {
			maxCut = pCh->cut;
		}
	}
	mCut = (int)((float)maxCut * 0.75f);
	::printf("coefs cut: %d .. %d -> %d\n", minCut, maxCut, mCut);

	size_t evalCoefsSize = mRotChansNum * (mCut + mCut - 1) * sizeof(float);
	hipHostMalloc(&mpEvalCoefsCPU, evalCoefsSize);
	if (!mpEvalCoefsCPU) return;
	float* pCoefs = mpEvalCoefsCPU;
	for (int i = 0; i < mRotChansNum; ++i) {
		ROT_CHANNEL* pCh = &mpRotChans[i];
		if (pCh->pCoefs) {
			*pCoefs++ = pCh->pCoefs[0];
			for (int j = 1; j < mCut; ++j) {
				*pCoefs++ = pCh->pCoefs[j];
			}
			for (int j = 1; j < mCut; ++j) {
				*pCoefs++ = pCh->pCoefs[j + (mCoefsNum / 2)];
			}
		}
	}
	hipMalloc(&mpEvalCoefsDev, evalCoefsSize);
	if (mpEvalCoefsDev) {
		hipMemcpy(mpEvalCoefsDev, mpEvalCoefsCPU, evalCoefsSize, hipMemcpyHostToDevice);
	}

	size_t evalResSize = mRotChansNum * sizeof(float);
	hipHostMalloc(&mpEvalResCPU, evalResSize);
	clear_res();
	hipMalloc(&mpEvalResDev, evalResSize);

	hipEventCreateWithFlags(&mEvt, hipEventDisableTiming);
}

void cMotion::unload() {
	hipDeviceSynchronize();
	hipEventDestroy(mEvt);
	if (mpEvalResDev) {
		hipFree(mpEvalResDev);
		mpEvalResDev = nullptr;
	}
	if (mpEvalResCPU) {
		hipHostFree(mpEvalResCPU);
		mpEvalResCPU = nullptr;
	}
	if (mpEvalCoefsDev) {
		hipFree(mpEvalCoefsDev);
		mpEvalCoefsDev = nullptr;
	}
	if (mpEvalCoefsCPU) {
		hipHostFree(mpEvalCoefsCPU);
		mpEvalCoefsCPU = nullptr;
	}
	if (mpRotChans) {
		for (int i = 0; i < mRotChansNum; ++i) {
			ROT_CHANNEL* pCh = &mpRotChans[i];
			if (pCh->pCoefs) {
				::free(pCh->pCoefs);
				pCh->pCoefs = nullptr;
			}
		}
		::free(mpRotChans);
		mpRotChans = nullptr;
		mRotChansNum = 0;
	}
	if (mpPosVecs) {
		::free(mpPosVecs);
		mpPosVecs = nullptr;
		mPosVecsNum = 0;
	}
	if (mpClip) {
		motClipUnload(mpClip);
		mpClip = nullptr;
	}
	mCut = 0;
	mCoefsNum = 0;
	mParamFactor = 0.0f;
}

void cMotion::eval_pos_vecs(float frame) {
	if (!mpClip) return;
	if (!mpPosVecs) return;
	int nfrm = mpClip->nfrm;
	int numNodes = mpClip->nnod;
	int idx = 0;
	for (int i = 0; i < numNodes; ++i) {
		int srcMask = mpClip->nodes[i].trk[POS].dataMask;
		int dataMask = mpClip->nodes[i].trk[POS].dataMask;
		if (srcMask || dataMask) {
			int fno = (int)frame;
			MOT_VEC v = motGetVec(mpClip, i, fno, POS);
			if (fno < nfrm - 1) {
				float t = frame - (float)fno;
				MOT_VEC v1 = motGetVec(mpClip, i, fno + 1, POS);
				for (int j = 0; j < 3; ++j) {
					v.v[j] += (v1.v[j] - v.v[j]) * t;
				}
			}
			mpPosVecs[idx] = v;
			++idx;
		}
	}
}

__host__ __device__
#if 0
void eval_sub(float* pRes, const float* pCoefs, float t, int n, int tid) {
	const float* pRe = &pCoefs[tid * (n + n - 1)];
	const float* pIm = &pRe[n];
	float res = pRe[0];
	for (int i = 1; i < n; ++i) {
		float r = t * (float)i;
		float re = pRe[i];
		float im = pIm[i - 1];
		res += re*cosf(r) + im*sinf(r);
	}
	pRes[tid] = res;
}
#else
// NR ed3: (5.4.6)
void eval_sub(float* pRes, const float* pCoefs, float t, int n, int tid) {
	const float* pRe = &pCoefs[tid * (n + n - 1)];
	const float* pIm = &pRe[n];
	float res = pRe[0];
	float r = t;
	float c = cosf(r);
	float s = sinf(r);
	float a = sinf(r*0.5f);
	a = 2.0f * a*a;
	float b = s;
	float re = pRe[1];
	float im = pIm[0];
	res += re*c + im*s;
	for (int i = 2; i < n; ++i) {
		float ci = c - (a*c + b*s);
		float si = s - (a*s - b*c);
		re = pRe[i];
		im = pIm[i - 1];
		res += re*ci + im*si;
		c = ci;
		s = si;
	}
	pRes[tid] = res;
}
#endif

void cMotion::eval_cpu(float frame) {
	float* pCoefs = mpEvalCoefsCPU;
	float* pRes = mpEvalResCPU;
	if (!pCoefs || !pRes) return;
	float t = frame_to_param(frame);
	int n = mRotChansNum;
//#pragma omp parallel for
	for (int i = 0; i < n; ++i) {
		eval_sub(pRes, pCoefs, t, mCut, i);
	}
	eval_pos_vecs(frame);
}

__global__ void eval_kernel(float* pRes, const float* pCoefs, float t, int n, int nres) {
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= nres) return;
	eval_sub(pRes, pCoefs, t, n, tid);
}

static int s_blkMin = 64;
static int s_blkMax = 128;

static int calc_thr_num(int nwk) {
	int n = (int)::log2(nwk) - 1;
	if (n < 0) n = 0;
	n = 1 << n;
	if (n < s_blkMin) n = s_blkMin;
	if (n > s_blkMax) n = s_blkMax;
	return n;
}

void cMotion::eval_dev(float frame) {
	float* pCoefs = mpEvalCoefsDev;
	float* pRes = mpEvalResDev;
	if (!pCoefs || !pRes) return;
	float t = frame_to_param(frame);
	int nch = mRotChansNum;
	int nthr = calc_thr_num(nch);
	int nblk = (nch + nthr - 1) / nthr;
	eval_kernel<<<nblk, nthr, 0, 0>>>(pRes, pCoefs, t, mCut, nch);
	hipMemcpyAsync(mpEvalResCPU, pRes, mRotChansNum * sizeof(float), hipMemcpyDeviceToHost, 0);
	hipEventRecord(mEvt, 0);
	eval_pos_vecs(frame);
	while (hipEventQuery(mEvt) == hipErrorNotReady) {}
}

static cMotion s_mot;

void init() {
	hipDeviceProp_t devProps;
	hipGetDeviceProperties(&devProps, 0);
	s_blkMax = devProps.maxThreadsPerBlock / 8;
	::printf("device: %s, compute %d.%d\n", devProps.name, devProps.major, devProps.minor);
	::printf("SM count = %d\n", devProps.multiProcessorCount);
	::printf("max thr/SM = %d\n", devProps.maxThreadsPerMultiProcessor);
	::printf("max thr/blk = %d\n", devProps.maxThreadsPerBlock);
	::printf("concurrent exec = %s\n", devProps.concurrentKernels ? "yes" : "no");
	::printf("\n");

	const char* pPath = "test.mclp";
	s_mot.load(pPath);
	::printf("#rot chans = %d\n", s_mot.get_nrot());
	::printf("#pos vecs = %d\n", s_mot.get_npos());
}


double res_l2() {
	double res = 0;
	int n = s_mot.get_nrot();
	float* p = s_mot.get_res_ptr();
	if (p) {
		for (int i = 0; i < n; ++i) {
			res += p[i] * p[i];
		}
		res = sqrt(res);
	}
	return res;
}


int main() {
	init();

	const int N = 1000;
	double cpuT = 0.0f;
	double devT = 0.0f;
	double devRes = 0.0;
	double cpuRes = 0.0;
	::printf("-----\n");

	for (int i = 0; i < N; ++i) {
		float frm = s_mot.get_nfrm() * float(i) / float(N);
		int64_t devT0 = timestamp();
		s_mot.eval_dev(frm);
		int64_t devT1 = timestamp();
		double devDT = (double)(devT1 - devT0);
		devT += devDT;
		devRes += res_l2();
	}
	devT /= N;
	::printf("dev res = %.1f\n", devRes);
	::printf("dev t = %.1f\n", devT);

	s_mot.clear_res();

	for (int i = 0; i < N; ++i) {
		float frm = s_mot.get_nfrm() * float(i) / float(N);
		int64_t cpuT0 = timestamp();
		s_mot.eval_cpu(frm);
		int64_t cpuT1 = timestamp();
		double cpuDT = (double)(cpuT1 - cpuT0);
		cpuT += cpuDT;
		cpuRes += res_l2();
	}
	cpuT /= N;
	::printf("cpu res = %.1f\n", cpuRes);
	::printf("cpu t = %.1f\n", cpuT);

	::printf("%f\n", cpuT / devT);

	return 0;
}
